/*
    Authors: Erkin Verbeek, Prabhat Bhootra
    Date: 12/2/2019

 *     *** with 3 x 3 patch ***
 *
	M = 10000
	N = 10000
	The elapsed time: 71.6 ms

    M = 30000
	N = 10000
	The elapsed time: 215.1 ms
	
	M = 20000
	N = 20000
	The elapsed time: 227.6 ms
	
 *     *** with 7 x 7 patch ***
 
	M = 10000
	N = 10000
	The elapsed time: 225.5 ms

	M = 30000
	N = 10000
	The elapsed time: 766.0 ms
 
	M = 20000
	N = 20000
	The elapsed time: 964.6 ms
	
	
*/
 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAX_VAL 255 // the max value for each pixel 
#define M 10000
#define N 10000

__global__ void blurKernel(unsigned char * in_img,
		unsigned char * out_img, int patchWidth) {
	
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	
	if (Col < N && Row < M) {
		int pixVal = 0;
		int pixels = 0;
	
	
		for (int blurRow = -patchWidth; blurRow < patchWidth + 1; blurRow++) {
			for (int blurCol = -patchWidth; blurCol < patchWidth + 1; blurCol++) {
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
			
				if (curRow > -1 && curRow < M && curCol > -1 && curCol < N) {
					pixVal += in_img[curRow*N + curCol];
					pixels++;
				}
			}
		}
		out_img[Row*N + Col] = (unsigned char) (pixVal/pixels);
	}
	
}


int main(int argc, char* argv[]) {
	int i;
	unsigned char * img;	// original IMG
	unsigned char * dev_img;	// original IMG
	unsigned char * dev_newImg;	// new IMG
	int size = M * N * sizeof(unsigned char);
	
	// dimensions of the grid and the block
	dim3 DimGrid((N-1)/16 + 1, (M-1)/16+1, 1);
	dim3 DimBlock(16, 16, 1);
	
	// allocates memory on device
	hipMalloc((void **)&dev_img, size);	// device space for original img
	hipMalloc((void **)&dev_newImg, size);	// device space for new img
	
	// allocate memory on host
	img = (unsigned char * ) malloc(size);
	
	srand(time(0)); // seed the random number generator
  
  	// we iterate through img and fill with random values
  	for (i = 0; i < M * N; i++) {
		img[i] = rand() % (MAX_VAL + 1);
		//printf("img[%d] = %d\n", i, img[i]);
  	}
  	printf("M = %d\nN = %d\n", M, N);
	
	hipMemcpy(dev_img, img, size, hipMemcpyHostToDevice);	// copy image from host to device
	
	float elapsedTime;		// timing stuff
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start,0);
	
	//launch dot() kernel with 1 block and N threads
	blurKernel<<<DimGrid, DimBlock>>>(dev_img, dev_newImg, 3);
	
	hipEventRecord(end, 0);	// stop timer
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsedTime,start,end);
	
	// copy device result back to the host copy of c
	
	hipMemcpy(img, dev_newImg, size, hipMemcpyDeviceToHost);	// copy new image back to host
	
	/*
	// write out new image so we can see if we did anything
  	for (i = 0; i < M * N; i++)
    	printf("newImg[%d] = %d\n", i, img[i]);
	*/
	
	// print elapsed time
	printf("The elapsed time: %3.1f ms\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(end);
	
	// free up heap space variables
	free(img);
	hipFree(dev_img);
	hipFree(dev_newImg);
	return 0;
}




