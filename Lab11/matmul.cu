#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width) {
	// Calculate the row index of the P element and M
	int Row = blockIdx.y*blockDim.y+ threadIdx.y;

	// Calculate the column index of P and N
	int Col = blockIdx.x*blockDim.x+ threadIdx.x;

	if ((Row < Width) && (Col < Width)) {
		float Pvalue= 0.0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < Width; ++k) {
			Pvalue += M[Row*Width+k]*N[k*Width+Col];
		}
		//printf("%f\n", Pvalue);
		P[Row*Width+Col] = Pvalue;
	}
}

int main (int argc, char *argv[]) {
	int i, j;
	int k = atoi(argv[1]);
	int *dev_k;
	float *m, *n, *p;
	float *dev_m, *dev_n, *dev_p;

	m = (float *) malloc(k*k*sizeof(float));
 	n = (float *) malloc(k*k*sizeof(float));
	p = (float *) malloc(k*k*sizeof(float));

	//malloc  k variable in device

	for (i = 0; i < k; i++) {
		for (j = 0; j < k; j++){
			m[i*k + j] = 1.0;
			n[i*k + j] = 1.0;
		}
	}
	hipMalloc ((void**) &dev_m, k*k*sizeof(float));
	hipMalloc ((void**) &dev_n, k*k*sizeof(float));
	hipMalloc ((void**) &dev_p, k*k*sizeof(float));
	hipMalloc ((void**) &dev_k, sizeof(int));

	hipMemcpy(dev_m, m, k*k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_n, n, k*k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&dev_k, k, sizeof(int), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	dim3 DimGrid((k-1)/16 + 1, (k-1)/16+1, 1);
	dim3 DimBlock(16, 16, 1);
	MatrixMulKernel<<<DimGrid, DimBlock>>>(dev_m, dev_n, dev_p, dev_k);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start, stop);
	printf("The elapsed time: %3.1f ms\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(p, dev_p, k*k*sizeof(float), hipMemcpyDeviceToHost);

	for (i = 0; i < k; i++) {
		for (j = 0; j < k; j++){
			printf("%f\n", p[i*k + j]);
		}
	}
	
	free(m);
	free(n);
	free(p);
	hipFree(dev_m);
	hipFree(dev_n);
	hipFree(dev_p);
}
